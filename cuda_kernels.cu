#include <hip/hip_runtime.h>
//#include <cuda_gl_interop.h>
#include <math.h>
#include <stdio.h>


// Kernel to animate sphere positions
extern "C" __global__ void animateSpheresKernel(float* data, int n, float time) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Animate each sphere in a circle (make movement obvious)
        float x = data[3 * idx + 0];
        float y = data[3 * idx + 1];
        float z = data[3 * idx + 2];
        data[3 * idx + 0] = x + 0.10f * cosf(time + idx); // Increased offset
        data[3 * idx + 1] = y + 0.10f * sinf(time + idx);
        data[3 * idx + 2] = z;
    }
}

// Host launcher for main.cpp
extern "C" void launchAnimateSpheresKernel(float* d_data, int n, float time) {
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    animateSpheresKernel<<<numBlocks, blockSize>>>(d_data, n, time);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    err = hipGetLastError();
	//printf("Inside launchAnimateSpheresKernel, after synchronize\n");
    if (err != hipSuccess) {
        printf("CUDA post-sync error: %s\n", hipGetErrorString(err));
    }
}
